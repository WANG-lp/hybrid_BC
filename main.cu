#include <iostream>
#include <iomanip>
#include <cstdlib>

#include "parse.h"
#include "sequential.h"
#include "util.cuh"
#include "kernels.cuh"

int main(int argc, char *argv[])
{
	program_options op = parse_arguments(argc,argv);
	int max_threads_per_block, number_of_SMs;
	choose_device(max_threads_per_block,number_of_SMs,op);
	
	graph g = parse(op.infile);
	graph g_out;
	int left_vertices = 0;
	if(op.one_deg_reduce){
		bool finish = reduce_1_degree_vertices(&g, &g_out);
		while (!finish) {
			finish = reduce_1_degree_vertices(&g_out, &g_out);
		}
		for (int i = 0; i < g.n; i++) {
			if (g_out.R[i + 1] - g_out.R[i] > 0) {
				left_vertices++;
			}
		}
		std::cout << "\tDeleted " << g.n - left_vertices << " vertices\n";
		std::cout << "\t1 degree vertices percent: " << (g.n - left_vertices) * 100 / (float) g.n << "%\n";
	}

	std::cout << "Number of nodes: " << g.n << std::endl;
	std::cout << "Number of edges: " << g.m << std::endl;

	//If we're approximating, choose source vertices at random
	std::set<int> source_vertices;
	if(op.approx)
	{
		op.k = g.n * op.ratio;
		if(op.k > g.n || op.k < 1)
		{
			op.k = g.n;
		}
		srand(0x4D5A);
		while(source_vertices.size() < op.k)
		{
			int temp_source = rand() % g.n;
			source_vertices.insert(temp_source);
		}
		std::cout << "vertices number: " << op.k << std::endl;
	}

	hipEvent_t start,end;
	float CPU_time;
	std::vector<float> bc;
	if(op.verify) //Only run CPU code if verifying
	{
		start_clock(start,end);
		bc = bc_cpu(g,source_vertices);
		CPU_time = end_clock(start,end);
	}

	float GPU_time;
	std::vector<float> bc_g;
	start_clock(start,end);
	if(op.one_deg_reduce){
		bc_g = bc_gpu(g_out, max_threads_per_block, number_of_SMs, op, source_vertices, op.one_deg_reduce, g_out.weight);
	}else {
		bc_g = bc_gpu(g, max_threads_per_block, number_of_SMs, op, source_vertices, op.one_deg_reduce, g.weight);
	}
	GPU_time = end_clock(start,end);

	if(op.verify)
	{
		verify(g,bc,bc_g);
	}
	if(op.printBCscores)
	{
		g.print_BC_scores(bc_g,op.scorefile);
	}

	std::cout << std::setprecision(9);
	if(op.verify)
	{
		std::cout << "Time for CPU Algorithm: " << CPU_time << " s" << std::endl;
	}
	std::cout << "Time for GPU Algorithm: " << GPU_time << " s" << std::endl;
	
	delete[] g.R;
	delete[] g.C;
	delete[] g.F;

	return 0;
}
