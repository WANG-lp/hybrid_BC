#include "hip/hip_runtime.h"
#include "util.cuh"
#include <queue>

//Note: Times are returned in seconds
void start_clock(hipEvent_t &start, hipEvent_t &end)
{
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&end));
	checkCudaErrors(hipEventRecord(start,0));
}

float end_clock(hipEvent_t &start, hipEvent_t &end)
{
	float time;
	checkCudaErrors(hipEventRecord(end,0));
	checkCudaErrors(hipEventSynchronize(end));
	checkCudaErrors(hipEventElapsedTime(&time,start,end));
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(end));

	return time/(float)1000;
}

program_options parse_arguments(int argc, char *argv[])
{
	program_options op;
	int c;

	static struct option long_options[] =
	{
		{"device",required_argument,0,'d'},
		{"help",no_argument,0,'h'},
		{"infile",required_argument,0,'i'},
		{"approx",required_argument,0,'k'},
		{"printscores",optional_argument,0,'p'},
		{"verify",no_argument,0,'v'},
		{0,0,0,0} //Terminate with null
	};

	int option_index = 0;

	while((c = getopt_long(argc,argv,"d:hi:k:p::v::r",long_options,&option_index)) != -1)
	{
		switch(c)
		{
			case 'd':
				op.device = atoi(optarg);
			break;

			case 'h':
				std::cout << "Usage: " << argv[0] << " -i <input graph file> [-v verify GPU calculation] [-p <output file> print BC scores] [-d <device ID> choose GPU (starting from 0)]" << std::endl;	
			exit(0);

			case 'i':
				op.infile = optarg;
			break;

			case 'k':{
				op.approx = true;
				int r = 0;
                for(int i = 0; i < strlen(optarg) -1; i++){
                    r = r * 10 + (optarg[i] - '0');
                }
                op.ratio = r / 100.0;
                std::cout << "ratio: " << op.ratio<<std::endl;
            }
			break;

			case 'p':
				op.printBCscores = true;
				op.scorefile = optarg;
			break;

			case 'v':
				op.verify = true;
			break;
            case 'r':
                op.one_deg_reduce = true;
                break;
			
			case '?': //Invalid argument: getopt will print the error msg itself
				
			exit(-1);

			default: //Fatal error
				std::cerr << "Fatal error parsing command line arguments. Terminating." << std::endl;
			exit(-1);

		}
	}

	if(op.infile == NULL)
	{
		std::cerr << "Command line error: Input graph file is required. Use the -i switch." << std::endl;
	}

	return op;
}

void choose_device(int &max_threads_per_block, int &number_of_SMs, program_options op)
{
	int count;
	checkCudaErrors(hipGetDeviceCount(&count));
	hipDeviceProp_t prop;

	if(op.device == -1)
	{
		int maxcc=0, bestdev=0;
		for(int i=0; i<count; i++)
		{
			checkCudaErrors(hipGetDeviceProperties(&prop,i));
			if((prop.major + 0.1*prop.minor) > maxcc)
			{
				maxcc = prop.major + 0.1*prop.minor;
				bestdev = i;
			}	
		}

		checkCudaErrors(hipSetDevice(bestdev));
		checkCudaErrors(hipGetDeviceProperties(&prop,bestdev));
	}
	else if((op.device < -1) || (op.device >= count))
	{
		std::cerr << "Invalid device argument. Valid devices on this machine range from 0 through " << count-1 << "." << std::endl;
		exit(-1);
	}
	else
	{
		checkCudaErrors(hipSetDevice(op.device));
		checkCudaErrors(hipGetDeviceProperties(&prop,op.device));
	}

	std::cout << "Chosen Device: " << prop.name << std::endl;
	std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
	std::cout << "Number of Streaming Multiprocessors: " << prop.multiProcessorCount << std::endl;
	std::cout << "Size of Global Memory: " << prop.totalGlobalMem/(float)(1024*1024*1024) << " GB" << std::endl << std::endl;

	max_threads_per_block = prop.maxThreadsPerBlock;
	number_of_SMs = prop.multiProcessorCount;
}

void verify(graph g, const std::vector<float> bc_cpu, const std::vector<float> bc_gpu)
{
	double error = 0;
	double max_error = 0;
	for(int i=0; i<g.n; i++)
	{
		double current_error = abs(bc_cpu[i] - bc_gpu[i]);
		error += current_error*current_error;
		if(current_error > max_error)
		{
			max_error = current_error;
		}
	}
	error = error/(float)g.n;
	error = sqrt(error);
	std::cout << "RMS Error: " << error << std::endl;
	std::cout << "Maximum error: " << max_error << std::endl;
}

bool reduce_1_degree_vertices(graph *in_g, graph *out_g) {
    out_g->total_comp = find_components_size(in_g);

    if (out_g->which_components == NULL) {
        out_g->R = new int[in_g->n + 1];
        out_g->F = new int[in_g->m * 2];
        out_g->C = new int[in_g->m * 2];
        out_g->weight = new int[in_g->n];
        std::fill_n(out_g->weight, in_g->n, 1);
        out_g->bc = new int[in_g->n];
        std::memset(out_g->bc, 0, in_g->n * sizeof(int));
        out_g->components_sizes = in_g->components_sizes;
        out_g->which_components = in_g->which_components;
        out_g->n = in_g->n;
        out_g->m = in_g->m;
    }

    int *R = new int[in_g->n + 1];
    int *F = new int[in_g->m * 2];
    int *C = new int[in_g->m * 2];

    std::memcpy(R, in_g->R, sizeof(int) * (in_g->n + 1));
    std::memcpy(F, in_g->F, sizeof(int) * (in_g->m * 2));
    std::memcpy(C, in_g->C, sizeof(int) * (in_g->m * 2));

    std::set<std::pair<int, int> > deleted;

    bool finish = true;
#ifdef DEBUG
    std::cout << "\tCSR INDEX ARRAY:\n\t\t";
    for (int i = 0; i < in_g->n; i++) {
        std::cout << R[i] << '\t';
    }
    std::cout << std::endl;
#endif

    for (int i = 0; i < in_g->n; i++) {
        if (R[i + 1] - R[i] == 1) {
            int v = C[R[i]];
            if (deleted.find(std::make_pair(i, v)) != deleted.end() ||
                deleted.find(std::make_pair(v, i)) != deleted.end())
                continue;
            finish = false;

            out_g->bc[i] += (out_g->weight[i] - 1) *
                            (out_g->components_sizes[out_g->which_components[i]] - out_g->weight[i]);
            out_g->bc[v] += (out_g->components_sizes[out_g->which_components[i]] - 1 - out_g->weight[i]) *
                            (out_g->weight[i]);
            out_g->weight[v] += out_g->weight[i];
//            out_g->bc[v] += 2 * (out_g->components_sizes[out_g->which_components[v]] -
//                                out_g->weight[v] - 1);
            out_g->which_components[i] = out_g->total_comp++;
            out_g->m--;
            //un-directed edge
            deleted.insert(std::make_pair(i, v));
            deleted.insert(std::make_pair(v, i));
        }
    }

    int r_index = 0;
    //int m = 0;
    for (int i = 0; i < in_g->n; i++) {
        out_g->R[i] = r_index;
        for (int j = R[i]; j < R[i + 1]; j++) {
            if (deleted.find(std::make_pair(i, C[j])) == deleted.end() &&
                deleted.find(std::make_pair(C[j], i)) == deleted.end()) {
                out_g->C[r_index] = C[j];
                out_g->F[r_index++] = i;
            }
        }
    }
    //std::cout << r_index << std::endl;
    out_g->R[in_g->n] = r_index;


    delete[] R;
    delete[] F;
    delete[] C;
    return finish;
}

int find_components_size(graph *g) {
    if (g->which_components != NULL)
        return g->total_comp;

    g->which_components = new int[g->n];

    std::vector<int> components_sizes(g->n, 0);

    std::vector<bool> vis(g->n, false);


    int total_components = 0;

    for (int i = 0; i < g->n; i++) {
        if (!vis[i]) {
            std::queue<int> Q;
            Q.push(i);
            vis[i] = true;
            components_sizes[total_components] = 1;
            g->which_components[i] = total_components;
            while (!Q.empty()) {
                int v = Q.front();
                Q.pop();
                for (int j = g->R[v]; j < g->R[v + 1]; j++) {
                    int u = g->C[j];
                    if (!vis[u]) {
                        vis[u] = true;
                        Q.push(u);
                        components_sizes[total_components]++;
                        g->which_components[u] = total_components;
                    }
                }
            }
            total_components++;
        }

    }
    g->components_sizes = new int[total_components];
    for (int i = 0; i < total_components; i++) {
        g->components_sizes[i] = components_sizes[i];
    }

    std::cout << "\tTotal components: " << total_components << "\n";
    return total_components;
}

